#include "hip/hip_runtime.h"
#include "ransac.hh"


// CUDA Kernel for calculating distances and checking inliers
__global__ void calculate_distances(
    const nova::Vec3f* points,
    std::size_t points_size,
    nova::Vec3f S0,
    float r,
    float threshold,
    float* distances,
    bool* is_inliers,
    unsigned int* inliers_count
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= points_size) return;

    // Calculate distance
    nova::Vec3f tmp_v3 = points[idx] - S0;
    nova::Vec2f tmp_v2 = {tmp_v3.x(), tmp_v3.y()};
    float dist = fabs(tmp_v2.length() - r);

    distances[idx] = dist;
    is_inliers[idx] = dist < threshold ? 1 : 0;

    // Count inliers using atomic addition
    if (is_inliers[idx]) {
        atomicAdd(inliers_count, 1);
    }
}

auto calculate_RANSAC_diffs_cuda(
    const nova::Vec3f* points,
    std::size_t points_size,
    const nova::Vec4f& cylinder,
    float threshold
)
        -> RANSAC_diffs_cuda
{
    const nova::Vec3f S0 { cylinder.x(), cylinder.y(), cylinder.z() };
    const float r = cylinder.w();

    // Allocate memory for device and host
    nova::Vec3f* d_points;
    float* d_distances;
    bool* d_is_inliers;
    unsigned int* d_inliers_count;
    unsigned int h_inliers_count = 0;

    hipMalloc(&d_points, points_size * sizeof(nova::Vec3f));
    hipMalloc(&d_distances, points_size * sizeof(float));
    hipMalloc(&d_is_inliers, points_size * sizeof(bool));
    hipMalloc(&d_inliers_count, sizeof(unsigned int));

    hipMemcpy(d_points, points, points_size * sizeof(nova::Vec3f), hipMemcpyHostToDevice);
    hipMemcpy(d_inliers_count, &h_inliers_count, sizeof(unsigned int), hipMemcpyHostToDevice);

    int blockSize = 256; // You can adjust the block size for optimization
    int numBlocks = (points_size + blockSize - 1) / blockSize;

    // Launch CUDA kernel
    calculate_distances<<<numBlocks, blockSize>>>(d_points, points_size, S0, r, threshold, d_distances, d_is_inliers, d_inliers_count);

    // Copy back results to host
    std::vector<float> distances(points_size);
    std::vector<char> is_inliers(points_size);
    hipMemcpy(distances.data(), d_distances, points_size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(is_inliers.data(), d_is_inliers, points_size * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(&h_inliers_count, d_inliers_count, sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_points);
    hipFree(d_distances);
    hipFree(d_is_inliers);
    hipFree(d_inliers_count);

    return {
        .num_inliers = h_inliers_count,
        .distances = distances,
        .is_inliers = is_inliers,
    };
}
